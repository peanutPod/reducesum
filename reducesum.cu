#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

//nvidia reduce pdf url :https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf

// Parallel Reduction: Interleaved Addressing
extern __global__ void reduce0(int *g_idata, int *g_odata);
//0.26ms
__global__ void reduce0(int *g_idata, int *g_odata) {
    extern __shared__ int sdata[];
    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    sdata[tid] = g_idata[i];
    __syncthreads();
    // do reduction in shared mem
    for(unsigned int s=1; s < blockDim.x; s *= 2) {
        if (tid % (2*s) == 0) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}


//0.26
//Reduction #2: Interleaved Addressing
extern __global__ void reduce1(int *g_idata, int *g_odata);
__global__ void reduce1(int *g_idata, int *g_odata) {
extern __shared__ int sdata[];
// each thread loads one element from global to shared mem
unsigned int tid = threadIdx.x;
unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
sdata[tid] = g_idata[i];
__syncthreads();
// do reduction in shared mem
//problem: highly divergent warps are very inefficient,and % operator is slow
for (unsigned int s=1; s < blockDim.x; s *= 2) {
if (tid % (2*s) == 0) {
sdata[tid] += sdata[tid + s];
}

__syncthreads();
}
// write result for this block to global mem
if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}


//0.24
//Parallel Reduction: Interleaved Addressing
extern __global__ void reduce2(int *g_idata, int *g_odata);
__global__ void reduce2(int *g_idata, int *g_odata) {
extern __shared__ int sdata[];
// each thread loads one element from global to shared mem
unsigned int tid = threadIdx.x;
unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
sdata[tid] = g_idata[i];
__syncthreads();
// do reduction in shared mem
//problem:shared memory bank conflicts
for (unsigned int s=1; s < blockDim.x; s *= 2) {
int index = 2 * s * tid;
if (index < blockDim.x) {
sdata[index] += sdata[index + s];
}
__syncthreads();
}
// write result for this block to global mem
if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}


//0.235 Parallel Reduction: Sequential Addressing
extern __global__ void reduce3(int *g_idata, int *g_odata);
__global__ void reduce3(int *g_idata, int *g_odata) {
extern __shared__ int sdata[];
// each thread loads one element from global to shared mem
unsigned int tid = threadIdx.x;
unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
sdata[tid] = g_idata[i];
__syncthreads();
// do reduction in shared mem
// sequential addressing is conflict free
//problem:Half of the threads are idle on first loop iteration!

for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
    if (tid < s) {
        sdata[tid] += sdata[tid + s];
    }
    __syncthreads();

}

// write result for this block to global mem
if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

//0.235 Reduction #4: First Add During Load
//Instruction Bottleneck
extern __global__ void reduce4(int *g_idata, int *g_odata);
__global__ void reduce4(int *g_idata, int *g_odata) {
extern __shared__ int sdata[];

//perform first level of reduction,
//reading from global memory, writing to shared memory
unsigned int tid = threadIdx.x;
unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
sdata[tid] = g_idata[i] + g_idata[i+blockDim.x];
__syncthreads();

// do reduction in shared mem
// sequential addressing is conflict free
//problem:Half of the threads are idle on first loop iteration!

for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
    if (tid < s) {
        sdata[tid] += sdata[tid + s];
    }
    __syncthreads();

}

// write result for this block to global mem
if (tid == 0) g_odata[blockIdx.x] = sdata[0];

}


//unroll loops
/*
Unrolling the Last Warp
As reduction proceeds, # “active” threads decreases
When s <= 32, we have only one warp left
Instructions are SIMD synchronous within a warp
That means when s <= 32:
We don’t need to __syncthreads()
We don’t need “if (tid < s)” because it doesn’t save any
work
Let’s unroll the last 6 iterations of the inner loop
*/
//important:for this to be correct ,we must use the "volatile" keyword!
__device__ void warpReduce (volatile int*sdata,int tid) {
    sdata[tid] += sdata[tid + 32];
    sdata[tid] += sdata[tid + 16];
    sdata[tid] += sdata[tid + 8];
    sdata[tid] += sdata[tid + 4];
    sdata[tid] += sdata[tid + 2];
    sdata[tid] += sdata[tid + 1];
}


/*
Note: This saves useless work in all warps, not just the last one!
Without unrolling, all warps execute every iteration of the for loop and if statement 
*/
extern __global__ void reduce5(int *g_idata, int *g_odata);
__global__ void reduce5(int *g_idata, int *g_odata) {
extern __shared__ int sdata[];

//perform first level of reduction,
//reading from global memory, writing to shared memory
unsigned int tid = threadIdx.x;
unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
sdata[tid] = g_idata[i] + g_idata[i+blockDim.x];
__syncthreads();

// do reduction in shared mem
// sequential addressing is conflict free
//problem:Half of the threads are idle on first loop iteration!

for (unsigned int s=blockDim.x/2; s>32; s>>=1) {
    if (tid < s) {
        sdata[tid] += sdata[tid + s];
    }
    __syncthreads();

}
if (tid < 32) warpReduce(sdata,tid);

// write result for this block to global mem
if (tid == 0) g_odata[blockIdx.x] = sdata[0];

}



template <unsigned int blockSize>
__device__ void warpReduce (volatile int*sdata,int tid) {
if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
}


template <unsigned int blockSize>
extern __global__ void reduce6(int *g_idata, int *g_odata);

template <unsigned int blockSize>
__global__ void reduce6(int *g_idata, int *g_odata) {
extern __shared__ int sdata[];

//perform first level of reduction,
//reading from global memory, writing to shared memory
unsigned int tid = threadIdx.x;
unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
sdata[tid] = g_idata[i] + g_idata[i+blockDim.x];
__syncthreads();

// do reduction in shared mem
// sequential addressing is conflict free
//problem:Half of the threads are idle on first loop iteration!

for (unsigned int s=blockDim.x/2; s>32; s>>=1) {
    if (blockSize >=512){
        if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads();
    }
    if (blockSize >=256){
        if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads();
    }
    if (blockSize >=128){
        if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads();
    }
}
if (tid < 32) warpReduce<blockSize>(sdata,tid);

// write result for this block to global mem
if (tid == 0) g_odata[blockIdx.x] = sdata[0];

}




//final kernel
template <unsigned int blockSize>
extern __global__ void reduce7(int *g_idata, int *g_odata,int n);

/*
Combine sequential and parallel reduction
    Each thread loads and sums multiple elements into
shared memory
    Tree-based reduction in shared memory
Brent’s theorem says each thread should sum
O(log n) elements
    i.e. 1024 or 2048 elements per block vs. 256
In my experience, beneficial to push it even further
    Possibly better latency hiding with more work per thread
    More threads per block reduces levels in tree of recursive
kernel invocations
    High kernel launch overhead in last levels with few blocks
On G80, best perf with 64-256 blocks of 128 threads
1   024-4096 elements per thread
*/
template <unsigned int blockSize>
__global__ void reduce7(int *g_idata, int *g_odata,int n ) {
extern __shared__ int sdata[];

//perform first level of reduction,
//reading from global memory, writing to shared memory
unsigned int tid = threadIdx.x;
unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
unsigned int gridSize = blockSize * 2 * gridDim.x;
sdata[tid] = 0;

while(i<n){
    sdata[tid] += g_idata[i] + g_idata[i+blockSize];
    //Note:gridSize loop stride to maintain coalescing!
    i += gridSize;
}
__syncthreads();

// do reduction in shared mem
// sequential addressing is conflict free
//problem:Half of the threads are idle on first loop iteration!

for (unsigned int s=blockDim.x/2; s>32; s>>=1) {
    if (blockSize >=512){
        if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads();
    }
    if (blockSize >=256){
        if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads();
    }
    if (blockSize >=128){
        if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads();
    }
}
if (tid < 32) warpReduce<blockSize>(sdata,tid);

// write result for this block to global mem
if (tid == 0) g_odata[blockIdx.x] = sdata[0];

}



void test_reduce(int size) {
    // Allocate host memory
    int *h_idata = (int*)malloc(size * sizeof(int));
    int *h_odata = (int*)malloc((size/1024) * sizeof(int));

    // Initialize input data
    for (int i = 0; i < size; i++) {
        h_idata[i] = 1;
    }

    // Allocate device memory
    int *d_idata, *d_odata;
    hipMalloc(&d_idata, size * sizeof(int));
    hipMalloc(&d_odata, (size/1024) * sizeof(int));

    // Copy input data to device
    hipMemcpy(d_idata, h_idata, size * sizeof(int), hipMemcpyHostToDevice);

    // Setup execution parameters
    int threads = 1024;
    int blocks = 512;

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record start event
    hipEventRecord(start);

    // Launch kernel
    // reduce5<<<blocks, threads, threads*sizeof(int)>>>(d_idata, d_odata);

    //reduce6 
    switch (threads)
        {
        case 512:
        reduce7<512><<< blocks, threads, threads*sizeof(int) >>>(d_idata, d_odata,size); break;
        case 256:
        reduce7<256><<< blocks, threads, threads*sizeof(int) >>>(d_idata, d_odata,size); break;
        case 128:
        reduce7<128><<< blocks, threads, threads*sizeof(int) >>>(d_idata, d_odata,size); break;
        case 64:
        reduce7< 64><<< blocks, threads, threads*sizeof(int) >>>(d_idata, d_odata,size); break;
        case 32:
        reduce7< 32><<< blocks, threads, threads*sizeof(int) >>>(d_idata, d_odata,size); break;
        case 16:
        reduce7< 16><<< blocks, threads, threads*sizeof(int) >>>(d_idata, d_odata,size); break;
        case 8:
        reduce7< 8><<< blocks, threads, threads*sizeof(int) >>>(d_idata, d_odata,size); break;
        case 4:
        reduce7< 4><<< blocks, threads, threads*sizeof(int) >>>(d_idata, d_odata,size); break;
        case 2:
        reduce7< 2><<< blocks, threads, threads*sizeof(int) >>>(d_idata, d_odata,size); break;
        case 1:
        reduce7< 1><<< blocks, threads, threads*sizeof(int) >>>(d_idata, d_odata,size); break;
        }

    // Record stop event
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Copy result back to host
    hipMemcpy(h_odata, d_odata, (size/1024) * sizeof(int), hipMemcpyDeviceToHost);

    // Verify result
    printf("Kernel execution time: %f ms\n", milliseconds);
    printf("First block result: %d\n", h_odata[0]);

    // Cleanup
    hipFree(d_idata);
    hipFree(d_odata);
    free(h_idata);
    free(h_odata);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

int main() {
    int size = 4096 * 1024; // 1M elements
    test_reduce(size);
    return 0;
}